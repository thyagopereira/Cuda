// This is a paralell version of a vector sum
#include "hip/hip_runtime.h"

#include <stdio.h>

int *a, *b, *c; // Host data

// Execute vector sum in paralell
__global__ void vecAdd(int* a, int* b, int* c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// Code to run in cpu
int main(){
    hipDeviceReset();
    int *d_a, *d_b, *d_c;
    int N = 256; 
    int size = N * sizeof(int);

    a = (int*)malloc(size);
    b = (int*)malloc(size); //Espaço de memoriade um array
    c = (int*)malloc(size); 

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    for(int i = 0; i < N; i++){
        a[i] = i; b[i] = i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    vecAdd<<<1,N>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Resultado da soma: \n");
    for(int i = 0; i < N; i++){
        printf("%d \n", c[i]);
    }
    
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}