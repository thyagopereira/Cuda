#include "hip/hip_runtime.h"
 
#include <stdio.h>
#include<unistd.h>


const int N = 256;

__global__  void produtoEscalar(int* a, int b, int* c){
    int i = threadIdx.x ;
    c[i] = a[i] * b;
}

int main(){
    hipDeviceReset();
    int *a, b, *c, *d_a, *d_c;

    int aSize = N * sizeof(int);
    a = (int*)malloc(aSize);
    c = (int*)malloc(aSize);

    hipMalloc((void**)&d_a,aSize);
    hipMalloc((void**)&d_c,aSize);

    b = 5;
    for(int i = 0; i < N; i++){
        a[i] = i;
    }

    hipMemcpy(d_a, a, aSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, aSize, hipMemcpyHostToDevice);

    produtoEscalar<<<1, N>>>(d_a, b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, aSize, hipMemcpyDeviceToHost);

    printf("Resultado escalar: \n");
    for(int i = 0; i < N; i++){
        printf("%d \n", c[i]);
    }

    hipFree(d_a); hipFree(d_c);
    return 0;
}